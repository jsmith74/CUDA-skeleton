#include "hip/hip_runtime.h"
#include "CUDAfunc.h"

__global__ void addCUDA( int *a, int *b, int *c ){

    *c = *a + *b;


}

void CUDAOffloader::getDeviceProperties(){

    int count;
    hipGetDeviceCount(&count);
    std::cout << "Number of Devices: " << count << std::endl << std::endl;

    hipDeviceProp_t Prop;
    hipGetDeviceProperties(&Prop,0);
    std::cout << Prop.name << std::endl;
    std::cout << "Amount of memory on device in bytes: " << Prop.totalGlobalMem << std::endl;
    std::cout << "The Maximum amount of shared memory a single block may use in bytes: " << Prop.sharedMemPerBlock << std::endl;
    std::cout << "Max threads per block: " << Prop.maxThreadsPerBlock << std::endl;
     //more stats you don't understand yet

    return;

}

void CUDAOffloader::add(int& c,int a,int b){

    int *dev_a, *dev_b, *dev_c;

    hipMalloc( (void**)&dev_a,sizeof(int) );
    hipMalloc( (void**)&dev_b,sizeof(int) );
    hipMalloc( (void**)&dev_c,sizeof(int) );

    hipMemcpy(dev_a,&a,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(dev_b,&b,sizeof(int),hipMemcpyHostToDevice);

    addCUDA<<<1,1>>>( dev_a, dev_b, dev_c );

    hipMemcpy( &c, dev_c, sizeof(int), hipMemcpyDeviceToHost  );

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return;

}


CUDAOffloader::CUDAOffloader(){


}
