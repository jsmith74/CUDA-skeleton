#include "hip/hip_runtime.h"
#include "CUDAfunc.h"

__global__ void addCUDA( int *a, int *b, int *c ){

    *c = *a + *b;


}

void CUDAOffloader::getDeviceProperties(){



    return;

}

void CUDAOffloader::add(int& c,int a,int b){

    int *dev_a, *dev_b, *dev_c;

    hipMalloc( (void**)&dev_a,sizeof(int) );
    hipMalloc( (void**)&dev_b,sizeof(int) );
    hipMalloc( (void**)&dev_c,sizeof(int) );

    hipMemcpy(dev_a,&a,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(dev_b,&b,sizeof(int),hipMemcpyHostToDevice);

    addCUDA<<<1,1>>>( dev_a, dev_b, dev_c );

    hipMemcpy( &c, dev_c, sizeof(int), hipMemcpyDeviceToHost  );

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return;

}


CUDAOffloader::CUDAOffloader(){


}
