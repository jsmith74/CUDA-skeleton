#include "hip/hip_runtime.h"
#include "CUDAfunc.h"

#define BLOCKS_PER_GRID 350

#define THREADS_PER_BLOCK 1024

__global__ void addCUDA( int *a, int *b, int *c ){

    *c = *a + *b;


}

__device__ __host__ double normC(thrust::complex<double>& c){

    return c.real() * c.real() + c.imag() * c.imag();

}

__global__ void dot(thrust::complex<double> *c,double *result){

//    if(threadIdx.x==0 && blockIdx.x==0){
//
//        for(int i=0;i<10;i++) printf("%d\t%d\n",c[i].real(),c[i].imag());
//
//        //assert(1>2);
//
//    }

    __shared__ double cache[THREADS_PER_BLOCK];

    int cacheIndex = threadIdx.x;

    double temp = 0.0;

    for(int k=0;k<100;k++) for(int j=0;j<10;j++) temp += normC( c[j] );

    for(int k=0;k<100;k++) for(int j=0;j<10;j++) temp -= normC( c[j] );

    for(int k=0;k<100;k++) for(int j=0;j<10;j++) temp += normC( c[j] );

    for(int k=0;k<100;k++) for(int j=0;j<10;j++) temp -= normC( c[j] );

    for(int k=0;k<100;k++) for(int j=0;j<10;j++) temp += normC( c[j] );

    for(int k=0;k<100;k++) for(int j=0;j<10;j++) temp -= normC( c[j] );

    for(int k=0;k<100;k++) for(int j=0;j<10;j++) temp += normC( c[j] );

    for(int k=0;k<100;k++) for(int j=0;j<10;j++) temp -= normC( c[j] );

    for(int k=0;k<100;k++) for(int j=0;j<10;j++) temp += normC( c[j] );

    for(int k=0;k<100;k++) for(int j=0;j<10;j++) temp -= normC( c[j] );

    for(int k=0;k<100;k++) for(int j=0;j<10;j++) temp += normC( c[j] );

    cache[cacheIndex] = temp;

    int i = blockDim.x / 2;

    __syncthreads();

    // this kind of reduction requires that number of threads is a power of 2

    while(i != 0){

        if(cacheIndex < i){

            cache[cacheIndex] += cache[cacheIndex + i];

        }

        __syncthreads();

        i /= 2;

    }

    if(cacheIndex == 0) result[blockIdx.x] = cache[0];

}

double CUDAOffloader::sendDataToDeviceAndCompute(){

    thrust::complex<double> *c,*cDev;

    c = (thrust::complex<double>*)malloc(10*sizeof(thrust::complex<double>));

    thrust::complex<double> I(0.0,1.0);

    for(int i=0;i<10;i++) c[i] = 1.0 * i + ( i + 0.5 ) * I;

    for(int i=0;i<10;i++) std::cout << "test: " << normC(c[i]) << std::endl;

    double* resultDev;

    double* result;

    result = (double*)malloc(BLOCKS_PER_GRID*sizeof(double));

    hipMalloc ( (void**)&cDev,10*sizeof(thrust::complex<double>) );

    hipMalloc ( (void**)&resultDev, BLOCKS_PER_GRID * sizeof(double) );

    hipMemcpy ( cDev, c, 10*sizeof(thrust::complex<double>), hipMemcpyHostToDevice );

    dot<<<BLOCKS_PER_GRID,THREADS_PER_BLOCK>>>(cDev,resultDev);

    hipMemcpy ( result, resultDev, BLOCKS_PER_GRID * sizeof(double), hipMemcpyDeviceToHost);

    double output = 0.0;

    for(int i=0;i<BLOCKS_PER_GRID;i++) output += result[i];

    hipFree ( cDev );

    hipFree ( resultDev );

    free(c);

    free(result);

    return output;

}

void CUDAOffloader::getDeviceProperties(){

    int count;
    hipGetDeviceCount(&count);
    std::cout << "Number of Devices: " << count << std::endl << std::endl;

    hipDeviceProp_t Prop;
    hipGetDeviceProperties(&Prop,0);
    std::cout << Prop.name << std::endl;
    std::cout << "Amount of memory on device in bytes: " << Prop.totalGlobalMem << std::endl;
    std::cout << "The Maximum amount of shared memory a single block may use in bytes: " << Prop.sharedMemPerBlock << std::endl;
    std::cout << "Max threads per block: " << Prop.maxThreadsPerBlock << std::endl;
    std::cout << "Number of 32-bit registers per block: " << Prop.regsPerBlock << std::endl;
    std::cout << "Maximum Number of blocks along x,y, and z:\n";
    std::cout << Prop.maxGridSize[0] << "\t" << Prop.maxGridSize[1] << "\t" << Prop.maxGridSize[2] << std::endl;
     //more stats you don't understand yet

    return;

}

void CUDAOffloader::add(int& c,int a,int b){

    int *dev_a, *dev_b, *dev_c;

    hipMalloc( (void**)&dev_a,sizeof(int) );
    hipMalloc( (void**)&dev_b,sizeof(int) );
    hipMalloc( (void**)&dev_c,sizeof(int) );

    hipMemcpy(dev_a,&a,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(dev_b,&b,sizeof(int),hipMemcpyHostToDevice);

    addCUDA<<<1,1>>>( dev_a, dev_b, dev_c );

    hipMemcpy( &c, dev_c, sizeof(int), hipMemcpyDeviceToHost  );

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return;

}


CUDAOffloader::CUDAOffloader(){



}
