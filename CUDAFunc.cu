#include "hip/hip_runtime.h"
#include "CUDAfunc.h"

#define BLOCKS_PER_GRID 350

#define THREADS_PER_BLOCK 1024

__device__ bool next_permutation(int* __first, int* __last);
__device__ void reverse(int* __first, int* __last);
__device__ inline double cNorm(thrust::complex<double>& c);
__device__ inline void iter_swap(int* __a, int* __b);

__global__ void reduce(thrust::complex<double> *Udev,double result[]){

    double output = 0.0;
    int k = 0;

    int cacheIndex = threadIdx.x;

    __shared__ double cache[THREADS_PER_BLOCK];

    int i[10];

    for(int j=0;j<10;j++) i[j] = j;

    do{

        output += cNorm(Udev[ k % 10 ]);
        output -= cNorm(Udev[ k % 10 ]);
        k++;

    } while(next_permutation( i,i+10) );

    cache[cacheIndex] = output;

    __syncthreads();

    int ii = blockDim.x/2;

    while(ii != 0){

        if(cacheIndex < ii){

            cache[cacheIndex] += cache[cacheIndex + ii];

        }

        __syncthreads();

        ii /= 2;

    }

    if(cacheIndex == 0) result[blockIdx.x] = cache[0];

}

CUDAOffloader::CUDAOffloader(){

}

double CUDAOffloader::sendDataandCompute(thrust::complex<double> U[]){

    thrust::complex<double> *Udev;

    double *resultDev;
    double *result;

    hipMalloc( (void**)&Udev, 10 * sizeof(thrust::complex<double>) );

    hipMalloc( (void**)&resultDev, BLOCKS_PER_GRID*sizeof(double) );

    hipMemcpy( Udev, U, 10*sizeof(thrust::complex<double>), hipMemcpyHostToDevice );

    reduce<<<BLOCKS_PER_GRID,THREADS_PER_BLOCK>>>(Udev, resultDev);

    result = (double*)malloc( BLOCKS_PER_GRID * sizeof(double) );

    hipMemcpy( result, resultDev, BLOCKS_PER_GRID*sizeof(double), hipMemcpyDeviceToHost );

    double output = 0.0;

    for(int i=0;i<BLOCKS_PER_GRID;i++){

        output += result[i];

    }

    hipFree( Udev );

    hipFree( resultDev );

    free( result );

    return output;

}

__device__ inline double cNorm(thrust::complex<double>& c){

    return c.real() * c.real() + c.imag() * c.imag();

}

__device__ inline void iter_swap(int* __a, int* __b) {
  int __tmp = *__a;
  *__a = *__b;
  *__b = __tmp;
}

__device__ void reverse(int* __first, int* __last) {

  while (true)
    if (__first == __last || __first == --__last)
      return;
    else{
      iter_swap(__first++, __last);
    }
}

__device__ bool next_permutation(int* __first, int* __last) {

  if (__first == __last)
    return false;
  int* __i = __first;
  ++__i;
  if (__i == __last)
    return false;
  __i = __last;
  --__i;

  for(;;) {
    int* __ii = __i;
    --__i;
    if (*__i < *__ii) {
      int* __j = __last;
      while (!(*__i < *--__j))
        {}
    iter_swap(__i, __j);
      reverse(__ii, __last);
      return true;
    }
    if (__i == __first) {
      reverse(__first, __last);
      return false;
    }
  }

}
